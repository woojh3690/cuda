#include "hip/hip_runtime.h"
#include "GPUACC.cuh"
#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include "hipblas.h"
#include <stdio.h>
#include <stdlib.h>

GPUACC::GPUACC(void)
{

}

GPUACC::~GPUACC(void)
{

}

__global__ void MatrixMulKernel(float* Md, float* Nd, float* Pd, int Width)
{
	//2D Thread ID
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	//Pvalue stores the Pd element that is computed by the thread
	float Pvalue = 0;

	for (int k = 0; k < Width; ++k)
	{
		float Mdelement = Md[ty * Width + k];
		float Ndelement = Nd[k * Width + tx];
		Pvalue += Mdelement * Ndelement;
	}

	//Write the matrix to device memory each thread writes one element
	Pd[ty * Width + tx] = Pvalue;
}

void GPUACC::MatrixMultiplication(float* M, float* N, float* P, int Width)
{

	int size = Width * Width * sizeof(float);
	float* Md;
	float* Nd;
	float* Pd;

	//Transfer M and N to device memory
	hipMalloc((void**)& Md, size);
	hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
	hipMalloc((void**)& Nd, size);
	hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);

	//Allocate P on the device
	hipMalloc((void**)& Pd, size);

	//Kernel invocation code - to be shown later
	//Setup the executioin configuration
	dim3 dimBlock(Width, Width);
	dim3 dimGrid(1, 1);

	//Launch the device computation threads!
	MatrixMulKernel <<<dimBlock, dimGrid>>> (Md, Nd, Pd, Width);


	//Transfer P from device to host
	//hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);
	//Free dvice matrices
	hipFree(Md); hipFree(Nd); hipFree(Pd);
}