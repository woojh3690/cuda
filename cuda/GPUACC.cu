#include "hip/hip_runtime.h"
#include "GPUACC.cuh"

#define TILE_WIDTH 25

GPUACC::GPUACC(void)
{

}

GPUACC::~GPUACC(void)
{

}

__global__ void MatrixMulKernel(float* Md, float* Nd, float* Pd, int Width)
{
	__shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
	__shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;

	//Identify the row and column of the Pd element to work on
	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH + tx;

	//printf("Row : %d,  Col : %d\n", Row, Col);

	float Pvalue = 0;
	//Loop over the Md and Nd tiles required to compute the Pd element
	for (int m = 0; m < Width / TILE_WIDTH; m++)
	{
		//Collaborative loading of Md and Nd tiles into shared memory
		Mds[ty][tx] = Md[Row * Width + (m * TILE_WIDTH + tx)];
		Nds[ty][tx] = Nd[(m * TILE_WIDTH + ty) * Width + Col];
		__syncthreads();

		for (int k = 0; k < TILE_WIDTH; k++)
			Pvalue += Mds[ty][k] * Nds[k][tx];
		__syncthreads();
	}

	Pd[Row * Width + Col] = Pvalue;

	////2D Thread ID
	//int tx = threadIdx.x;
	//int ty = threadIdx.y;

	////Pvalue stores the Pd element that is computed by the thread
	//float Pvalue = 0;

	//for (int k = 0; k < Width; k++)
	//{
	//	float Mdelement = Md[ty * Width + k];
	//	float Ndelement = Nd[k * Width + tx];
	//	//printf("��ǥ : %d, %d\n", ty * Width + k, k * Width + tx);
	//	Pvalue += Mdelement * Ndelement;
	//}

	////Write the matrix to device memory each thread writes one element
	//Pd[ty * Width + tx] = Pvalue;
}

void GPUACC::MatrixMultiplication(float* M, float* N, float* P, int Width)
{

	int size = Width * Width * sizeof(float);
	float* Md;
	float* Nd;
	float* Pd;

	//Transfer M and N to device memory
	hipMalloc((void**)& Md, size);
	hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
	hipMalloc((void**)& Nd, size);
	hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);

	//Allocate P on the device
	hipMalloc((void**)& Pd, size);

	//Kernel invocation code - to be shown later
	//Setup the executioin configuration
	dim3 dimGrid(2, 2);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);

	//Launch the device computation threads!
	MatrixMulKernel <<<dimGrid,dimBlock>>> (Md, Nd, Pd, Width);


	//Transfer P from device to host
	hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);
	//Free dvice matrices
	hipFree(Md); hipFree(Nd); hipFree(Pd);
}