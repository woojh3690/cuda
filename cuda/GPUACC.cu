#include "hip/hip_runtime.h"
#include "GPUACC.cuh"

#define TILE_WIDTH 7

GPUACC::GPUACC(void)
{

}

GPUACC::~GPUACC(void)
{

}

__global__ void MatrixMulKernel(float* Md, float* Nd, float* Pd, int Width)
{
	__shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
	__shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;

	//Identify the row and column of the Pd element to work on
	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH + tx;

	//printf("Row : %d,  Col : %d\n", Row, Col);

	float Pvalue = 0;
	//Loop over the Md and Nd tiles required to compute the Pd element
	for (int m = 0; m < Width / TILE_WIDTH; m++)
	{
		//Collaborative loading of Md and Nd tiles into shared memory
		Mds[ty][tx] = Md[Row * Width + (m * TILE_WIDTH + tx)];
		Nds[ty][tx] = Nd[(m * TILE_WIDTH + ty) * Width + Col];
		__syncthreads();

		for (int k = 0; k < TILE_WIDTH; k++)
			Pvalue += Mds[ty][k] * Nds[k][tx];
		__syncthreads();
	}

	Pd[Row * Width + Col] = Pvalue;
}

double GPUACC::MatrixMultiplication(float* M, float* N, float* P, int Width)
{

	int size = Width * Width * sizeof(float);
	float* Md;
	float* Nd;
	float* Pd;

	//Transfer M and N to device memory
	hipMalloc((void**)& Md, size);
	hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
	hipMalloc((void**)& Nd, size);
	hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);

	//Allocate P on the device
	hipMalloc((void**)& Pd, size);

	//Kernel invocation code - to be shown later
	//Setup the executioin configuration
	dim3 dimGrid(3, 3);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);

	//Launch the device computation threads!
	clock_t start = clock();
	for (int i = 0; i < 1000; i++) {
		MatrixMulKernel <<<dimGrid, dimBlock >>> (Md, Nd, Pd, Width);
	}
	clock_t end = clock();

	//Transfer P from device to host
	hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);
	//Free dvice matrices
	hipFree(Md); hipFree(Nd); hipFree(Pd);

	return (double)end - start;
}